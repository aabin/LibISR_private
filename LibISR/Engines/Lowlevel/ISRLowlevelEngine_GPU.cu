#include "hip/hip_runtime.h"
#include "ISRLowlevelEngine_GPU.h"
#include "ISRLowlevelEngine_DA.h"

#include "../../../LibISRUtils/IOUtil.h"

using namespace LibISR;
using namespace LibISR::Engine;
using namespace LibISR::Objects;

__global__ void subsampleImageRGBDImage_device(const Vector4f *imageData_in, Vector2i oldDims, Vector4f *imageData_out, Vector2i newDims);

__global__ void prepareAlignedRGBDData_device(Vector4f* rgbd_out, const short *depth_in, const Vector4u *rgb_in, const Vector2i imgSize, Matrix3f H, Vector3f T);

__global__ void preparePointCloudFromAlignedRGBDImage_device(Vector4f* ptcloud_out, Vector4f* inimg, float* histogram, Vector4f intrinsic, Vector4i boundingbox, Vector2i imgSize, int histBins);

__global__ void computepfImageFromHistogram_device(Vector4u* inimg, float* histogram, Vector2i imgSize, int histBins);

__global__ void convertNormalizedRGB_device(Vector4u* inimg, Vector4u* outimg, Vector2i imgSize);

//////////////////////////////////////////////////////////////////////////
// host functions
//////////////////////////////////////////////////////////////////////////

void LibISR::Engine::ISRLowlevelEngine_GPU::subsampleImageRGBDImage(ISRFloat4Image *outimg, ISRFloat4Image *inimg)
{
	Vector2i oldDims = inimg->noDims;
	Vector2i newDims; newDims.x = inimg->noDims.x / 2; newDims.y = inimg->noDims.y / 2;
	outimg->ChangeDims(newDims);

	const Vector4f *imageData_in = inimg->GetData(true);
	Vector4f *imageData_out = outimg->GetData(true);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)newDims.x / (float)blockSize.x), (int)ceil((float)newDims.y / (float)blockSize.y));

	subsampleImageRGBDImage_device << <gridSize, blockSize >> >(imageData_in, oldDims, imageData_out, newDims);
}

void LibISR::Engine::ISRLowlevelEngine_GPU::prepareAlignedRGBDData(ISRFloat4Image *outimg, ISRShortImage *raw_depth_in, ISRUChar4Image *rgb_in, Objects::ISRExHomography *home)
{
	int w = raw_depth_in->noDims.width;
	int h = raw_depth_in->noDims.height;

	short* depth_in_ptr = raw_depth_in->GetData(true);
	Vector4u* rgb_in_ptr = rgb_in->GetData(true);
	Vector4f* rgbd_out_ptr = outimg->GetData(true);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)w / (float)blockSize.x), (int)ceil((float)h / (float)blockSize.y));

	prepareAlignedRGBDData_device << <gridSize, blockSize >> >(rgbd_out_ptr, depth_in_ptr, rgb_in_ptr, raw_depth_in->noDims, home->H, home->T);
}

void LibISR::Engine::ISRLowlevelEngine_GPU::preparePointCloudFromAlignedRGBDImage(ISRFloat4Image *ptcloud_out, ISRFloat4Image *inimg, Objects::ISRHistogram *histogram, const Vector4f &intrinsic, const Vector4i &boundingbox)
{
	if (inimg->noDims != ptcloud_out->noDims) ptcloud_out->ChangeDims(inimg->noDims);
	
	int w = inimg->noDims.width;
	int h = inimg->noDims.height;

	int noBins = histogram->noBins;

	Vector4f *inimg_ptr = inimg->GetData(true);
	Vector4f* ptcloud_ptr = ptcloud_out->GetData(true);
	float* histogram_ptr = histogram->getPosteriorHistogram(true);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)w / (float)blockSize.x), (int)ceil((float)h / (float)blockSize.y));

	preparePointCloudFromAlignedRGBDImage_device << <gridSize, blockSize >> >(ptcloud_ptr, inimg_ptr, histogram_ptr, intrinsic, boundingbox, inimg->noDims, noBins);
}

void LibISR::Engine::ISRLowlevelEngine_GPU::computepfImageFromHistogram(ISRUChar4Image *rgb_in, Objects::ISRHistogram *histogram)
{
	
	int w = rgb_in->noDims.width;
	int h = rgb_in->noDims.height;

	int noBins = histogram->noBins;

	Vector4u *inimg_ptr = rgb_in->GetData(true);
	float* histogram_ptr = histogram->getPosteriorHistogram(true);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)w / (float)blockSize.x), (int)ceil((float)h / (float)blockSize.y));

	computepfImageFromHistogram_device << <gridSize, blockSize >> >(inimg_ptr, histogram_ptr, rgb_in->noDims, noBins);
	rgb_in->UpdateHostFromDevice();
}

void LibISR::Engine::ISRLowlevelEngine_GPU::convertNormalizedRGB(ISRUChar4Image* inrgb, ISRUChar4Image* outrgb)
{
	int w = inrgb->noDims.width;
	int h = inrgb->noDims.height;

	Vector4u *inimg_ptr = inrgb->GetData(true);
	Vector4u *outimg_ptr = outrgb->GetData(true);

	dim3 blockSize(16, 16);
	dim3 gridSize((int)ceil((float)w / (float)blockSize.x), (int)ceil((float)h / (float)blockSize.y));


	convertNormalizedRGB_device << <gridSize, blockSize >> >(inimg_ptr, outimg_ptr,outrgb->noDims);
	outrgb->UpdateHostFromDevice();
}


//////////////////////////////////////////////////////////////////////////
// device functions
//////////////////////////////////////////////////////////////////////////

__global__ void subsampleImageRGBDImage_device(const Vector4f *imageData_in, Vector2i oldDims, Vector4f *imageData_out, Vector2i newDims)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x > newDims.x - 1 || y > newDims.y - 1) return;

	filterSubsampleWithHoles(imageData_out, x, y, newDims, imageData_in, oldDims);
}

__global__ void prepareAlignedRGBDData_device(Vector4f* rgbd_out, const short *depth_in, const Vector4u *rgb_in, const Vector2i imgSize, Matrix3f H, Vector3f T)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x > imgSize.x - 1 || y > imgSize.y - 1) return;

	int idx = y * imgSize.x + x;
	ushort rawdepth = (ushort)depth_in[idx];
	float z = rawdepth == 65535 ? 0 : ((float)rawdepth) / 1000.0f;

	if (T.x == 0 && T.y == 0 && T.z == 0)
	{
		rgbd_out[idx].x = rgb_in[idx].r;
		rgbd_out[idx].y = rgb_in[idx].g;
		rgbd_out[idx].z = rgb_in[idx].b;
		rgbd_out[idx].w = z;
		return;
	}
	rgbd_out[idx].w = z;
	mapRGBDtoRGB(rgbd_out[idx], Vector3f(x*z, y*z, z), rgb_in, imgSize, H, T);
}

__global__ void preparePointCloudFromAlignedRGBDImage_device(Vector4f* ptcloud_out, Vector4f* inimg, float* histogram, Vector4f intrinsic, Vector4i boundingbox, Vector2i imgSize, int histBins)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x > imgSize.x - 1 || y > imgSize.y - 1) return;
	
	int idx = y * imgSize.x + x;

	if (x < boundingbox.x || x >= boundingbox.z || y < boundingbox.y || y >= boundingbox.w)
	{ 
		ptcloud_out[idx] = Vector4f(0, 0, 0, -1);
	}
	else
	{
		float z = inimg[idx].w;
		unprojectPtWithIntrinsic(intrinsic, Vector3f(x*z, y*z, z), ptcloud_out[idx]);

		ptcloud_out[idx].w = getPf(inimg[idx], histogram, histBins);
	}
}

__global__ void computepfImageFromHistogram_device(Vector4u* inimg, float* histogram, Vector2i imgSize, int histBins)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x > imgSize.x - 1 || y > imgSize.y - 1) return;

	int idx = y * imgSize.x + x;
	float pf = getPf(inimg[idx], histogram, histBins);

	if (pf>0.5f)
	{
		inimg[idx].r = 255;
		inimg[idx].g = 0;
		inimg[idx].b = 0;
	}
	else if (pf==0.5f)
	{
		inimg[idx].r = 0;
		inimg[idx].g = 0;
		inimg[idx].b = 255;
	}

}

__global__ void convertNormalizedRGB_device(Vector4u* inimg, Vector4u* outimg, Vector2i imgSize)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x, y = threadIdx.y + blockIdx.y * blockDim.y;
	if (x > imgSize.x - 1 || y > imgSize.y - 1) return;
	int idx = y * imgSize.x + x;

	float r, g, b, nm, nr, ng, nb;

	r = inimg[idx].r;
	g = inimg[idx].g;
	b = inimg[idx].b;

	if (r == 0, g == 0, b == 0) outimg[idx] = Vector4u((uchar)0);
	else
	{
		nm = 1 / sqrtf(r*r + g*g + b*b);
		nr = r*nm; ng = g*nm; nb = b*nm;
		outimg[idx].r = (uchar)(nr * 255);
		outimg[idx].g = (uchar)(ng * 255);
		outimg[idx].b = (uchar)(nb * 255);
	}
}

